#include "hip/hip_runtime.h"
#include <stdio.h>

#include "../../include/gpu/utils.h"

__global__ void average_rows_kernel(const int n, const int m,
                                    const int increment,
                                    const float* __restrict__ input,
                                    float* __restrict__ averages) {
  const int row = blockIdx.x * blockDim.x + threadIdx.x;
  if (row >= n)
    return;
  float        sum       = 0.0f;
  const float* row_start = input + row * increment;
  for (int j = 0; j < m; j++) {
    sum += row_start[j];
  }
  averages[row] = sum / (float) m;
}

__global__ void average_rows_kernel_parallel(const int n, const int m,
                                             const int increment,
                                             const float* __restrict__ input,
                                             float* __restrict__ averages) {
  extern __shared__ float sdata[];
  const int               row = blockIdx.x;
  if (row >= n)
    return;
  const int    tid        = threadIdx.x;
  const float* row_start  = input + row * increment;
  float        thread_sum = 0.0f;
  for (int j = tid; j < m; j += blockDim.x) {
    thread_sum += row_start[j];
  }
  sdata[tid] = thread_sum;
  __syncthreads();
  for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (tid < s) {
      sdata[tid] += sdata[tid + s];
    }
    __syncthreads();
  }
  if (tid == 0) {
    averages[row] = sdata[0] / (float) m;
  }
}

extern "C" int average_rows_gpu(const int n, const int m, const int increment,
                                const float* device_input, float* host_averages,
                                float* timing) {

  INIT();
  int         error_flag = event_error_flag;
  hipError_t last_error = hipSuccess;
  START();
  int       threadsPerBlock;
  int       numBlocks;
  bool      useParallelKernel;
  int       sharedMemSize           = 0;
  const int max_blocks_for_parallel = 1024;
  const int min_cols_for_parallel   = 64;
  if (n <= max_blocks_for_parallel && m >= min_cols_for_parallel) {
    threadsPerBlock   = 256;
    numBlocks         = n;
    useParallelKernel = true;
    sharedMemSize     = threadsPerBlock * sizeof(float);
  } else {
    threadsPerBlock   = 256;
    numBlocks         = (n + threadsPerBlock - 1) / threadsPerBlock;
    useParallelKernel = false;
    sharedMemSize     = 0;
  }
  END();
  START();
  float* device_averages = NULL;
  last_error = hipMalloc((void**) &device_averages, n * sizeof(float));
  if (last_error != hipSuccess) {
    fprintf(stderr, "ERROR: Failed to allocate device averages memory.\n");
    error_flag = 1;
    END();
    return error_flag;
  }
  END();

  // timing[2]: Transfer to...
  START();
  END(); // No host-to-device transferring needed

  START();
  if (!error_flag) {
    if (useParallelKernel) {
      average_rows_kernel_parallel<<<numBlocks, threadsPerBlock,
                                     sharedMemSize>>>(
          n, m, increment, device_input, device_averages);
    } else {
      average_rows_kernel<<<numBlocks, threadsPerBlock>>>(
          n, m, increment, device_input, device_averages);
    }
    last_error = hipGetLastError();
    if (last_error != hipSuccess) {
      fprintf(stderr, "ERROR: Averaging kernel launch failed.\n");
      error_flag = 1;
    } else {
      last_error = hipDeviceSynchronize();
      if (last_error != hipSuccess) {
        fprintf(stderr, "ERROR: Averaging kernel synchronization failed.\n");
        error_flag = 1;
      }
    }
  }
  END();
  START();
  if (!error_flag) {
    last_error = hipMemcpy(host_averages, device_averages, n * sizeof(float),
                            hipMemcpyDeviceToHost);
    if (last_error != hipSuccess) {
      fprintf(stderr, "ERROR: Failed to copy averages from device.\n");
      error_flag = 1;
    }
  }
  END();
  if (device_averages != NULL) {
    last_error = hipFree(device_averages);
    if (last_error != hipSuccess) {
      fprintf(stderr, "WARNING: Failed to free device averages memory.\n");
      if (error_flag == 0)
        error_flag = 1;
    }
  }
  COMPLETE();
  return error_flag;
}
