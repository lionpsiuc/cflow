
#include <hip/hip_runtime.h>
#include <stdio.h>

#define TIME_INIT()                                                            \
  hipEvent_t start;                                                           \
  hipEvent_t end;                                                             \
  hipEventCreate(&start);                                                     \
  hipEventCreate(&end);                                                       \
  int timing_index = 0

#define TIME_START() hipEventRecord(start)

#define TIME_END()                                                             \
  hipEventRecord(end);                                                        \
  if (timing != NULL) {                                                        \
    hipEventSynchronize(start);                                               \
    hipEventSynchronize(end);                                                 \
    hipEventElapsedTime(timing + timing_index, start, end);                   \
    timing[timing_index] /= 1000.0f;                                           \
  }                                                                            \
  timing_index++

#define TIME_FINISH()                                                          \
  hipEventDestroy(start);                                                     \
  hipEventDestroy(end);

// Initialise the grid on the GPU
__global__ void init_gpu(const int n, const int m, const int increment,
                         float* const grid) {
  const int i = blockIdx.y * blockDim.y + threadIdx.y;
  const int j = blockIdx.x * blockDim.x + threadIdx.x;

  // Check if the thread is within bounds
  if (i < n && j < m) {
    float col0 = 0.98f * (float) ((i + 1) * (i + 1)) / (float) (n * n);
    if (j == 0) {
      grid[i * increment + j]     = col0; // First column
      grid[i * increment + m + 0] = col0; // Set ghost column
    } else {

      // Interior points
      grid[i * increment + j] =
          col0 * ((float) (m - j) * (m - j) / (float) (m * m));

      // Moreover, set the other ghost column
      if (j == 1) {
        grid[i * increment + m + 1] = grid[i * increment + j];
      }
    }
  }
}

extern "C" void init_gpu_wrapper(float* host_grid, int n, int m,
                                 float* timing) {
  const int increment = m + 2;
  dim3      threadsPerBlock(16, 16);
  dim3      numBlocks((m + threadsPerBlock.x - 1) / threadsPerBlock.x,
                      (n + threadsPerBlock.y - 1) / threadsPerBlock.y);
  float*    device_grid;
  size_t    grid_size = n * increment * sizeof(float);
  TIME_INIT();

  // Allocate memory on the GPU
  TIME_START();
  hipMalloc((void**) &device_grid, grid_size);
  hipMemset(device_grid, 0, grid_size);
  TIME_END();

  // Run the kernel
  TIME_START();
  init_gpu<<<numBlocks, threadsPerBlock>>>(n, m, increment, device_grid);
  hipDeviceSynchronize();
  TIME_END();

  // Copy the result back to the host
  TIME_START();
  hipMemcpy(host_grid, device_grid, grid_size, hipMemcpyDeviceToHost);
  TIME_END();

  // Free GPU memory
  hipFree(device_grid);

  TIME_FINISH();
}
