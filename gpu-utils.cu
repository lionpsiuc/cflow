#include "gpu_utils.h"

/**
 * @brief Gets the index of the currently selected CUDA device; helper function
 *        used internally.
 *
 * @return Device index, or -1 on error.
 */
static int get_current_device_index(void) {
  int         device_index = -1;
  hipError_t error        = hipGetDevice(&device_index);
  if (error != hipSuccess) {
    return -1;
  }
  return device_index;
}

/**
 * @brief Gets the maximum shared memory per block usable by the currently
 *        selected device. Taken from your sample code.
 *
 * @return Maximum shared memory in bytes, or negative on error.
 */
int sm_per_block(void) {
  int device_index = get_current_device_index();
  if (device_index < 0) {
    return -1;
  }
  int         max_shared_bytes = -1;
  hipError_t error            = hipDeviceGetAttribute(
      &max_shared_bytes, hipDeviceAttributeSharedMemPerBlockOptin, device_index);
  if (error != hipSuccess ||
      max_shared_bytes <= 0) { // If hipDeviceAttributeSharedMemPerBlockOptin
                               // fails or isn't supported/meaningful
    error = hipDeviceGetAttribute(
        &max_shared_bytes, hipDeviceAttributeMaxSharedMemoryPerBlock, device_index);
  }
  if (error != hipSuccess) {
    return -1;
  }
  return max_shared_bytes;
}

/**
 * @brief Gets the compute capability of the currently selected device.
 *
 * @return Compute capability value, or negative on error.
 */
int current_cc(void) {
  int device_index = get_current_device_index();
  if (device_index < 0) {
    return -1;
  }
  int         major_cc = -1;
  hipError_t error    = hipDeviceGetAttribute(
      &major_cc, hipDeviceAttributeComputeCapabilityMajor, device_index);
  if (error != hipSuccess) {
    return -1;
  }
  int minor_cc = -1;
  error = hipDeviceGetAttribute(&minor_cc, hipDeviceAttributeComputeCapabilityMinor,
                                 device_index);
  if (error != hipSuccess) {
    return -1;
  }
  return major_cc * 100 + minor_cc * 10;
}
